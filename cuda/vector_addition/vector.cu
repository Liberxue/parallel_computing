#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>


#define N 512

void host_add (int *a, int *b,int *c){
  for (int idx = 0x00;idx < N; idx++)
    c[idx] = a[idx] + b[idx];
}

void fill_array(int *data) {
  for (int idx = 0x00;idx < N;idx++)
    data[idx] = idx;
}

void print_output(int *a,int *b ,int *c) {
  for(int idx = 0; idx < N; idx++)
    printf("\n %d + %d = %d",a[idx],b[idx],c[idx]);
}

__global__ void deivce_add(int *a ,int *b, int *c) {
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x]
}

int main(void) {
  int *a,*b,*c;
  int *d_a,*d_b,*d_c;
  int size = N * sizeof(int);
  a = (int *) malloc(size); fill_array(a);
  b = (int *) malloc(size); fill_array(b);
  c = (int *) malloc(size);
  // Alloc space for deivce
  hipMalloc((void *)&d_a,N * sizeof(int));
  hipMalloc((void *)&d_b,N * sizeof(int));
  hipMalloc((void *)&d_c,N * sizeof(int));
  // copy form host to deivce
  cudaMemcopy(d_a,a, N * sizeof(int),hipMemcpyHostToDevice);

  cudaMemcopy(d_b,b, N * sizeof(int),hipMemcpyHostToDevice);

  threads_pre_block = 8;
  no_of_blocks = N /threads_pre_block;

  deivce_add<<<no_of_blocks,threads_pre_block>>>(d_a,d_b,d_c);
  // copy result back to host
  hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyHostToDevice);

  // host_add(a,b,c);
  print_output(a,b,c);
  // free(a);free(b);free(c);
  hipFree(d_a);hipFree(d_b); hipFree(d_c);

  return 0x00;
}
