#include "hip/hip_runtime.h"
#include <math.h>
#include "hip/hip_runtime.h"
#include "kernel.h"

// declare the kernel function
__global__ void kernel_sum(const float* A, const float* B, float* C, int n_el);

// function which invokes the kernel
void sum(const float* A, const float* B, float* C, int n_el) {

  // declare the number of blocks per grid and the number of threads per block
  int threadsPerBlock,blocksPerGrid;

  // use 1 to 512 threads per block
  if (n_el<512){
    threadsPerBlock = n_el;
    blocksPerGrid   = 1;
  } else {
    threadsPerBlock = 512;
    blocksPerGrid   = ceil(double(n_el)/double(threadsPerBlock));
  }

  // invoke the kernel
  kernel_sum<<<blocksPerGrid,threadsPerBlock>>>(A, B, C, n_el);
}

// kernel
__global__ void kernel_sum(const float* A, const float* B, float* C, int n_el)
{
  // calculate the unique thread index
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  // perform tid-th elements addition
  if (tid < n_el) C[tid] = A[tid] + B[tid];
}
