#include <iostream>
#include <hip/hip_runtime.h>

__device__ int findLSB(unsigned int num) {
    if (num == 0) return -1;
    return __ffs(num) - 1; // Find the first set bit (LSB) and subtract 1 to get the index
}

__global__ void findLSBKernel(unsigned int* input, int* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = findLSB(input[idx]);
    }
}

int main() {
    int n = 5;
    unsigned int input[] = {0x80000000, 0x12345678, 0x00000001, 0xFEDCBA98, 0x01020304};
    unsigned int* dev_input;
    hipMalloc((void**)&dev_input, n * sizeof(unsigned int));
    hipMemcpy(dev_input, input, n * sizeof(unsigned int), hipMemcpyHostToDevice);

    int* dev_output;
    hipMalloc((void**)&dev_output, n * sizeof(int));

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    findLSBKernel<<<blocksPerGrid, threadsPerBlock>>>(dev_input, dev_output, n);

    int* output = new int[n];
    hipMemcpy(output, dev_output, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        std::cout << "Input: 0x" << std::hex << input[i] << ", LSB position: " << output[i] << std::endl;
    }

    delete[] output;
    hipFree(dev_input);
    hipFree(dev_output);

    return 0;
}
